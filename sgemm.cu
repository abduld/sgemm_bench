
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include "hipblas.h"
#define WA HA
#define WB HA
#define HB WA
#define WC WB
#define HC HA
#define index(i,j,ld) (((j)*(ld))+(i))

int  main (int argc, char** argv) {
  hipblasStatus_t status;
  int i,j;
  cublasInit();

  int HA =  atoi(argv[1]);
  float *A = (float*)malloc(HA*WA*sizeof(float));
  float *B = (float*)malloc(HB*WB*sizeof(float));
  float *C = (float*)malloc(HC*WC*sizeof(float));
  if (A == 0) {
    fprintf (stderr, "!!!! host memory allocation error (A)\n");
    return EXIT_FAILURE;
  }
  if (B == 0) {
    fprintf (stderr, "!!!! host memory allocation error (A)\n");
    return EXIT_FAILURE;
  }
  if (C == 0) {
    fprintf (stderr, "!!!! host memory allocation error (A)\n");
    return EXIT_FAILURE;
  }


  for (i=0;i<HA;i++)
    for (j=0;j<WA;j++)
      A[index(i,j,HA)] = (float) index(i,j,HA);
  for (i=0;i<HB;i++)
    for (j=0;j<WB;j++)
      B[index(i,j,HB)] = (float) index(i,j,HB);

  float* AA; float* BB; float* CC;

  /*ALLOCATE ON THE DEVICE*/
  status=cublasAlloc(HA*WA,sizeof(float),(void**)&AA);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf (stderr, "!!!! device memory allocation error (A)\n");
    return EXIT_FAILURE;
  }

  status=cublasAlloc(HB*WB,sizeof(float),(void**)&BB);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf (stderr, "!!!! device memory allocation error (A)\n");
    return EXIT_FAILURE;
  }

  status=cublasAlloc(HC*WC,sizeof(float),(void**)&CC);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf (stderr, "!!!! device memory allocation error (A)\n");
    return EXIT_FAILURE;
  }

  /*SET MATRIX*/
  status=hipblasSetMatrix(HA,WA,sizeof(float),A,HA,AA,HA);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf (stderr, "!!!! device memory allocation error (A)\n");
    return EXIT_FAILURE;
  }

  status=hipblasSetMatrix(HB,WB,sizeof(float),B,HB,BB,HB);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf (stderr, "!!!! device memory allocation error (A)\n");
    return EXIT_FAILURE;
  }

  /*KERNEL*/
  cublasSgemm('n','n',HA,WB,WA,1,AA,HA,BB,HB,0,CC,HC);
  cublasSgemm('n','t',HA,WB,WA,1,AA,HA,BB,HB,0,CC,HC);

  status = cublasGetError();
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf (stderr, "!!!! kernel execution error.\n");
    return EXIT_FAILURE;
  }
  hipblasGetMatrix(HC,WC,sizeof(float),CC,HC,C,HC);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf (stderr, "!!!! device read error (A)\n");
    return EXIT_FAILURE;
  }


  /* PERFORMANCE OUTPUT*/

  free( A );  free( B );  free ( C );
  status = cublasFree(AA);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf (stderr, "!!!! memory free error (A)\n");
    return EXIT_FAILURE;
  }
  status = cublasFree(BB);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf (stderr, "!!!! memory free error (B)\n");
    return EXIT_FAILURE;
  }
  status = cublasFree(CC);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf (stderr, "!!!! memory free error (C)\n");
    return EXIT_FAILURE;
  }

  /* Shutdown */
  status = cublasShutdown();
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf (stderr, "!!!! shutdown error (A)\n");
    return EXIT_FAILURE;
  }

  return EXIT_SUCCESS;
}

